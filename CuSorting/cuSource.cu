
#include "cuSource.h"
#include <hip/hip_runtime.h>

#include "GPU_Sorting_Functions.cuh"

void CuSource::sort() {
	switch (column_decide % 3)
	{
	case 0:
		break;
	case 1:
		// call sorting kernel with
		// d_paperIdWrapper
		switch (column_decide / 3)
		{
		case 2:
			odd_even_sort_scheme_paperid<<<NUM_BLOCK,WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
		default:
			break;
		}

		break;
	case 2:
		break;
	}

}

void CuSource::MemAllo(const char* file_name)
{
	Source::MemAllo(file_name);

	//hipMalloc((void **) d_schemeDataStructure, rows * sizeof(SchemeDataStructure));

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		paperIdWrapper = (struct PaperIdWrapper_Scheme *) malloc(rows * sizeof(struct PaperIdWrapper_Scheme));
		for (size_t i = 0; i < rows; i++) {
			paperIdWrapper[i].paper_id = paper_id[i];
			paperIdWrapper[i].classPtr = &schemeDataStructure[i];
		}
		hipMalloc((void **)d_paperIdWrapper, rows * sizeof(struct PaperIdWrapper_Scheme));
		hipMemcpy(d_paperIdWrapper, paperIdWrapper,
			rows * sizeof(struct PaperIdWrapper_Scheme),
			hipMemcpyHostToDevice
		);
		free(paperIdWrapper);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

}


void CuSource::MemFree()
{
	//hipFree(d_schemeDataStructure);

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		hipFree(d_paperIdWrapper);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	Source::MemFree();
}



void CuSource::postSorting()
{

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		hipMemcpy(paperIdWrapper, d_paperIdWrapper,
			rows * sizeof(struct PaperIdWrapper_Scheme),
			hipMemcpyDeviceToHost
		);
		for (size_t i = 0; i < rows; i++) {
			schemeDataStructure[i] = *paperIdWrapper[i].classPtr;
		}
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

}

void CuSource::write_file(const char * file_name, SchemeDataStructure * schemeDataStructure)
{
	FILE * p_file;
	std::string sorted_file_name(file_name);

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_sm_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");

	std::vector<std::string>::iterator iter;
	for (iter = headers.begin();
		iter != headers.end(); ++iter) {
		if (iter != headers.begin())
			printf_stream(p_file, ",");
		printf_stream(p_file, "%s", (*iter).c_str());
	}

	printf_stream(p_file, "\n");

	struct SchemeDSHolder schemeDSHolder;

	for (size_t i = 0; i < rows; i++)
	{

		schemeDataStructure[i].getValue(&schemeDSHolder);

		printf_stream(p_file, "%d,%s,%lld,%s,%s,%s,%d,%s,%d,%d,%s,%s,%d,%s,%s,%s,%s,%s,%s,%s,%s\n",
			schemeDSHolder.scheme_prog_code, schemeDSHolder.prog_name.c_str(), schemeDSHolder.scheme_id, schemeDSHolder.prog_sem_year.c_str(),
			schemeDSHolder.prepared_date.c_str(), schemeDSHolder.declared_date.c_str(), schemeDSHolder.institution_code, schemeDSHolder.institution_name.c_str(),
			schemeDSHolder.s_number, schemeDSHolder.paper_id, schemeDSHolder.paper_code.c_str(), schemeDSHolder.subject_name.c_str(), schemeDSHolder.credits,
			schemeDSHolder.type.c_str(), schemeDSHolder.exam.c_str(), schemeDSHolder.mode.c_str(), schemeDSHolder.kind.c_str(),
			schemeDSHolder.minor.c_str(), schemeDSHolder.major.c_str(), schemeDSHolder.max_marks.c_str(), schemeDSHolder.pass_marks.c_str())
			;

	}

	fclose(p_file);
}


