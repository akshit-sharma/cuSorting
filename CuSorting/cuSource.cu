
#include "cuSource.h"
#include <hip/hip_runtime.h>

#include "GPU_Sorting_Functions.cuh"

void CuSource::sort() {
	unsigned int num_blocks;
	switch (column_decide % 3)
	{
	case 0:
		break;
	case 1:
		// call sorting kernel with
		// d_paperIdWrapper
		switch ((column_decide-1) / 3)
		{
		case 0:
			quicksort_scheme_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
			break;
		case 1:
			shellsort_scheme_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
			break;
		case 2:
			num_blocks = static_cast<int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_scheme_paperid<<<num_blocks, WID_BLOCK>>>(d_paperIdWrapper, rows);
				gpuErrchk( hipPeekAtLastError() );
				gpuErrchk( hipDeviceSynchronize() );
			}
			checkArray << <static_cast<int>(rows / WID_BLOCK) + 1, WID_BLOCK >> > (d_paperIdWrapper, rows);
			break;
		default:
			break;
		}
		break;
	case 2:
		break;
	}

}

void CuSource::MemAllo(const char* file_name)
{
	Source::MemAllo(file_name);

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		paperIdWrapper = (struct PaperIdWrapper_Scheme *) malloc(rows * sizeof(struct PaperIdWrapper_Scheme));
		gpuErrchk(
			hipMalloc((void **)&d_paperIdWrapper, rows * sizeof(struct PaperIdWrapper_Scheme))
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}


}

void CuSource::preSorting()
{
	Source::preSorting();
	//hipMalloc((void **) d_schemeDataStructure, rows * sizeof(SchemeDataStructure));

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		for (size_t i = 0; i < rows; i++) {
			paperIdWrapper[i].paper_id = paper_id[i];
			paperIdWrapper[i].classPtr = &schemeDataStructure[i];

		//	printf_stream(stderr, "class add not same \n %p is before %p \n\n",
		//		&schemeDataStructure[i], paperIdWrapper[i].classPtr);
		}
		gpuErrchk(
		hipMemcpy(d_paperIdWrapper, paperIdWrapper,
			rows * sizeof(struct PaperIdWrapper_Scheme),
			hipMemcpyHostToDevice
		)
			);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

}


void CuSource::MemFree()
{
	//hipFree(d_schemeDataStructure);

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
		hipFree(d_paperIdWrapper)
		);
		free(paperIdWrapper);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	Source::MemFree();
}



void CuSource::postSorting()
{

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
		hipMemcpy(paperIdWrapper, d_paperIdWrapper,
			rows * sizeof(struct PaperIdWrapper_Scheme),
			hipMemcpyDeviceToHost
		));
		int paper_id_old;
		int paper_id_new;
		int wrongCount;
		paper_id_old = INT_MIN;
		wrongCount = 0;
		for (size_t i = 0; i < rows; i++) {
			paper_id_new = paperIdWrapper[i].paper_id;
			if (paper_id_new < paper_id_old) {
				printf_stream(stderr, "Array not sorted \n %d is before %d \n\n",
					paper_id_old, paper_id_new);
				}
			schemeDataStructure[i].setValue(paperIdWrapper[i].classPtr);
			if (paper_id_new != schemeDataStructure[i].getPaperId()) {
			//	printf_stream(stderr, "class add not same \n %p is before %p \n\n",
			//		&schemeDataStructure[i], paperIdWrapper[i].classPtr);
				wrongCount++;
			}
		}
		printf_stream(stdout, "Wrongly sorted elements %d \n", wrongCount);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

}

void CuSource::write_file(const char * file_name, SchemeDataStructure * schemeDataStructure)
{
	FILE * p_file;
	std::string sorted_file_name(file_name);

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_sm_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");

	std::vector<std::string>::iterator iter;
	for (iter = headers.begin();
		iter != headers.end(); ++iter) {
		if (iter != headers.begin())
			printf_stream(p_file, ",");
		printf_stream(p_file, "%s", (*iter).c_str());
	}

	printf_stream(p_file, "\n");

	struct SchemeDSHolder schemeDSHolder;

	for (size_t i = 0; i < rows; i++)
	{

		schemeDataStructure[i].getValue(&schemeDSHolder);

		printf_stream(p_file, "%d,%s,%lld,%s,%s,%s,%d,%s,%d,%d,%s,%s,%d,%s,%s,%s,%s,%s,%s,%s,%s\n",
			schemeDSHolder.scheme_prog_code, schemeDSHolder.prog_name.c_str(), schemeDSHolder.scheme_id, schemeDSHolder.prog_sem_year.c_str(),
			schemeDSHolder.prepared_date.c_str(), schemeDSHolder.declared_date.c_str(), schemeDSHolder.institution_code, schemeDSHolder.institution_name.c_str(),
			schemeDSHolder.s_number, schemeDSHolder.paper_id, schemeDSHolder.paper_code.c_str(), schemeDSHolder.subject_name.c_str(), schemeDSHolder.credits,
			schemeDSHolder.type.c_str(), schemeDSHolder.exam.c_str(), schemeDSHolder.mode.c_str(), schemeDSHolder.kind.c_str(),
			schemeDSHolder.minor.c_str(), schemeDSHolder.major.c_str(), schemeDSHolder.max_marks.c_str(), schemeDSHolder.pass_marks.c_str())
			;

	}

	fclose(p_file);
}


