
#include "cuSource.h"
#include <hip/hip_runtime.h>
#include "Debugger.h"

#include "GPU_Sorting_Functions.cuh"

void CuSource::sort() {

	unsigned int num_blocks;

	switch (column_decide % 3)
	{
	case 0:
		break;
	case 1:
		// call sorting kernel with
		// d_paperIdWrapper
		switch ((column_decide - 1) / 3)
		{
		case 0:
			//quicksort_scheme_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
			break;
		case 1:
			//shellsort_scheme_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
			break;
		case 2:
			num_blocks = static_cast<int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_int<<<num_blocks, WID_BLOCK>>>(d_int, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}
			break;
		default:
			break;
		}
		break;
	case 2:
		break;
	}

}

void CuSource::MemAllo(const char* file_name)
{
	Source::MemAllo(file_name);

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		gpuErrchk(
			hipMalloc((void **)&d_int, rows * sizeof(TYPE_PAPER_ID))
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}


}

void CuSource::preSorting()
{
	Source::preSorting();

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		gpuErrchk(
			hipMemcpy(d_int, paper_id,
				rows * sizeof(int),
				hipMemcpyHostToDevice
			)
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

}


void CuSource::MemFree()
{
	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipFree(d_int)
		);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	Source::MemFree();
}



void CuSource::postSorting()
{

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipMemcpy(paper_id, d_int,
				rows * sizeof(int),
				hipMemcpyDeviceToHost
			)
		);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

}

void CuSource::write_file(const char * file_name, SchemeDataStructure * schemeDataStructure)
{
	FILE * p_file;
	std::string sorted_file_name(file_name);
	int i_value;
	std::string s_value;

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_sm_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");
	
	for (size_t i = 0; i < rows; i++)
	{

		switch (column_decide % 3)
		{
		case 0:
			s_value = sorted_col_string[i];
			printf_stream(p_file, "%s\n", s_value.c_str());
			break;
		case 1:
			printf_stream(p_file, "%d\n", paper_id[i]);
			break;
		case 2:
			s_value = sorted_col_string[i];
			printf_stream(p_file, "%s\n", s_value.c_str());
			break;
		}

	}

	fclose(p_file);
}


