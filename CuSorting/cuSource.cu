
#include "cuSource.h"
#include <hip/hip_runtime.h>
#include "Debugger.h"

#include "GPU_Sorting_Functions.cuh"

void CuSource::sort() {

	unsigned int num_blocks;

	switch (column_decide % 3)
	{
	case 0:
		break;
	case 1:
		// call sorting kernel with
		// d_paperIdWrapper
		switch ((column_decide - 1) / 3)
		{
		case 0:
			quicksort_scheme_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
			break;
		case 1:
			shellsort_scheme_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
			break;
		case 2:
			num_blocks = static_cast<int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			//print_debug_specific
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_scheme_paperid<<<num_blocks, WID_BLOCK>>>(d_paperIdWrapper, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}
			//checkArray<<<static_cast<int>(rows / WID_BLOCK) + 1, WID_BLOCK>>>(d_paperIdWrapper, rows);
			break;
		default:
			break;
		}
		break;
	case 2:
		break;
	}

}

void CuSource::MemAllo(const char* file_name)
{
	Source::MemAllo(file_name);

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		paperIdWrapper = (struct PaperIdWrapper_Scheme *) malloc(rows * sizeof(struct PaperIdWrapper_Scheme));
		gpuErrchk(
			hipMalloc((void **)&d_paperIdWrapper, rows * sizeof(struct PaperIdWrapper_Scheme))
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}


}

void CuSource::preSorting()
{
	Source::preSorting();
	//hipMalloc((void **) d_schemeDataStructure, rows * sizeof(SchemeDataStructure));

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		for (size_t i = 0; i < rows; i++) {
			paperIdWrapper[i].paper_id = paper_id[i];
			paperIdWrapper[i].classPtr = &schemeDataStructure[i];

			//	printf_stream(stderr, "class add not same \n %p is before %p \n\n",
			//		&schemeDataStructure[i], paperIdWrapper[i].classPtr);
		}
		//copy_data(paperIdWrapper, rows);
		gpuErrchk(
			hipMemcpy(d_paperIdWrapper, paperIdWrapper,
				rows * sizeof(struct PaperIdWrapper_Scheme),
				hipMemcpyHostToDevice
			)
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

}


void CuSource::MemFree()
{
	//hipFree(d_schemeDataStructure);

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipFree(d_paperIdWrapper)
		);
		free(paperIdWrapper);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	Source::MemFree();
}



void CuSource::postSorting()
{

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipMemcpy(paperIdWrapper, d_paperIdWrapper,
				rows * sizeof(struct PaperIdWrapper_Scheme),
				hipMemcpyDeviceToHost
			));
		//print_debug_specific
		//check_data(paperIdWrapper);
		//remove_data();
		int paper_id_old;
		int paper_id_new;
		int wrongCount;
		int wrongCombination;
		SchemeDataStructure * tempSchemeDataStructure;
		paper_id_old = INT_MIN;
		wrongCount = 0;
		wrongCombination = 0;
		for (size_t i = 0; i < rows; i++) {
			paper_id_new = paperIdWrapper[i].paper_id;
			if (paper_id_new < paper_id_old) {
				wrongCount++;
			}
			tempSchemeDataStructure = paperIdWrapper[i].classPtr;
			if (paper_id_new != tempSchemeDataStructure->getPaperId()) {
//				printf("%d %d not same", paper_id_new, tempSchemeDataStructure->getPaperId());
			}
			auto originalValue = tempSchemeDataStructure->getOriginalValue();
			schemeDataStructure[i].setValue(&originalValue);
			if (paper_id_new != schemeDataStructure[i].getPaperId()) {
				wrongCombination++;
			}
			paper_id_old = paper_id_new;
		}
		printf_stream(stdout, "Wrongly sorted elements %d and combination %d \n", wrongCount, wrongCombination);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

}

void CuSource::write_file(const char * file_name, SchemeDataStructure * schemeDataStructure)
{
	FILE * p_file;
	std::string sorted_file_name(file_name);

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_sm_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");

	std::vector<std::string>::iterator iter;
	for (iter = headers.begin();
		iter != headers.end(); ++iter) {
		if (iter != headers.begin())
			printf_stream(p_file, ",");
		printf_stream(p_file, "%s", (*iter).c_str());
	}

	printf_stream(p_file, "\n");

	struct SchemeDSHolder schemeDSHolder;

	for (size_t i = 0; i < rows; i++)
	{

		schemeDataStructure[i].getValue(&schemeDSHolder);

		printf_stream(p_file, "%d,%s,%lld,%s,%s,%s,%d,%s,%d,%d,%s,%s,%d,%s,%s,%s,%s,%s,%s,%s,%s\n",
			schemeDSHolder.scheme_prog_code, schemeDSHolder.prog_name.c_str(), schemeDSHolder.scheme_id, schemeDSHolder.prog_sem_year.c_str(),
			schemeDSHolder.prepared_date.c_str(), schemeDSHolder.declared_date.c_str(), schemeDSHolder.institution_code, schemeDSHolder.institution_name.c_str(),
			schemeDSHolder.s_number, schemeDSHolder.paper_id, schemeDSHolder.paper_code.c_str(), schemeDSHolder.subject_name.c_str(), schemeDSHolder.credits,
			schemeDSHolder.type.c_str(), schemeDSHolder.exam.c_str(), schemeDSHolder.mode.c_str(), schemeDSHolder.kind.c_str(),
			schemeDSHolder.minor.c_str(), schemeDSHolder.major.c_str(), schemeDSHolder.max_marks.c_str(), schemeDSHolder.pass_marks.c_str())
			;

	}

	fclose(p_file);
}


