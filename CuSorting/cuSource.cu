#include "hip/hip_runtime.h"

#include "cuSource.h"
#include <hip/hip_runtime.h>

#include "time.h"

#include "GPU_Sorting_Functions.cuh"

double CuSource::sort() {

	hipEvent_t start, stop;
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	unsigned int num_blocks;
	unsigned int num_threads_per_block;
	bool sorted;
	int pre_val;
	size_t arr_size, num_arr;
	size_t left, right;
	size_t gpu_loop;

	switch (column_decide % 3)
	{
	case 0:
		break;
	case 1:
		// call sorting kernel with
		// d_paperIdWrapper
		switch ((column_decide - 1) / 3)
		{
		case 0:
			gpuErrchk(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));
			left = 0;
			right = static_cast<size_t>(rows - 1);
			quicksort_int<<< 1, 1>>>(d_int, left, right, 0);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
			break;
		case 1:			
			for (unsigned int i = rows/2; i > 0; i /= 2) {
				num_threads_per_block = rows;
				num_arr = i + 1;
				if ((num_threads_per_block - 1) / WID_BLOCK > 0) {
					num_blocks = (num_threads_per_block / WID_BLOCK) + 1;
					num_threads_per_block = WID_BLOCK;
				}
				arr_size = rows / i;
				
				//transfer data from d_int to d_xtra_int
				shellsort_int << <num_blocks, num_threads_per_block >> >(d_int, rows, num_arr-1, arr_size, d_xtra_int);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
				/*
				gpuErrchk(
					hipMemcpy(paper_id, d_xtra_int,
						rows * sizeof(int),
						hipMemcpyDeviceToHost
					)
				);
				printf_stream(stdout, "Xtra_Arr (Source.cu) Array size %d \n %5d", i, 0);
				for (unsigned int i = 0; i < rows; i++) {
					if(paper_id[i] == 0)
					printf_stream(stdout, " \n %5d %7d", i, paper_id[i]);

//					if ((i + 1) % arr_size == 0)
//						printf_stream(stdout, " \n %5d",(i+1)/arr_size);
				}
				printf_stream(stdout, " \n");
				*/

				//odd_even_sort_int_xtra(int * d_int, size_t maxLimit, size_t offset)
				gpu_loop = 1024;
				num_blocks = static_cast<unsigned int>(rows / WID_BLOCK) + 1;
				num_blocks = (num_blocks / 2) + 1;
				for (unsigned int i = 0; i < (arr_size/2)+1; i++) {
					odd_even_sort_int_xtra <<<num_blocks, WID_BLOCK >> >(d_xtra_int, rows, num_arr, arr_size);
					gpuErrchk(hipPeekAtLastError());
					gpuErrchk(hipDeviceSynchronize());
				}
				
				sorted = true;
				gpuErrchk(
					hipMemcpy(paper_id, d_xtra_int,
						rows * sizeof(int),
						hipMemcpyDeviceToHost
					)
				);
				pre_val = INT_MIN;
				for (unsigned int i = 0; i < rows; i++) {
					if (pre_val > paper_id[i]) {
						sorted = false;
						printf_stream(stdout, "Elements %7d(%7d) and %7d(%7d) not sorted",paper_id[i-1],i-1,paper_id[i],i);
						break;
					}
					pre_val = paper_id[i];

					if ((i + 1) % arr_size == 0)
						pre_val = INT_MIN;
				}
				if (!sorted) {
					printf_stream(stdout, "Xtra_Arr (Source.cu) Array size %d \n %5d", i, 0);
					for (unsigned int i = 0; i < rows; i++) {
						printf_stream(stdout, " %7d", paper_id[i]);

						if ((i + 1) % arr_size == 0)
							printf_stream(stdout, " \n %5d", (i + 1) / arr_size);
					}
					printf_stream(stdout, " \n");
				}
				
				num_threads_per_block = rows;
				if ((num_threads_per_block - 1) / WID_BLOCK > 0) {
					num_blocks = (num_threads_per_block / WID_BLOCK) + 1;
					num_threads_per_block = WID_BLOCK;
				}
				arr_size = rows / i;

				//transfer data from d_int to d_xtra_int
				if (i == 1) {
					gpuErrchk(
						hipMemcpy(d_int, d_xtra_int,
							rows * sizeof(int),
							hipMemcpyDeviceToDevice
						)
					);
				}
				else {
					shellsort_int_back << <num_blocks, num_threads_per_block >> > (d_int, rows, num_arr-1, arr_size, d_xtra_int);
					gpuErrchk(hipPeekAtLastError());
					gpuErrchk(hipDeviceSynchronize());
				}
			}
			break;
		case 2:
			gpu_loop = 1024;
			num_blocks = static_cast<unsigned int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_int<<<num_blocks, WID_BLOCK>>>(d_int, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}
			break;
		default:
			break;
		}
		break;
	case 2:
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return getTimeElapsed(start, stop);

}

double CuSource::MemAllo(const char* file_name)
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	c_start = startTimer();

	Source::MemAllo(file_name);

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		gpuErrchk(
			hipMalloc((void **)&d_int, rows * sizeof(TYPE_PAPER_ID))
		);
		gpuErrchk(
			hipMalloc((void **)&d_xtra_int, rows * sizeof(TYPE_PAPER_ID))
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end)+getTimeElapsed(start, stop);
}

double CuSource::preSorting()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	size_t i;

	c_start = startTimer();

	Source::preSorting();

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		gpuErrchk(
			hipMemcpy(d_int, paper_id,
				rows * sizeof(int),
				hipMemcpyHostToDevice
			)
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}


double CuSource::MemFree()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipFree(d_xtra_int)
		);
		gpuErrchk(
			hipFree(d_int)
		);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	c_start = startTimer();

	Source::MemFree();

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}



double CuSource::postSorting()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	int t_int;
	size_t i;

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipMemcpy(paper_id, d_int,
				rows * sizeof(int),
				hipMemcpyDeviceToHost
			)
		);
		/*
		t_int = paper_id[0];
		for (i = 0; i < rows; i++) {
			if (paper_id[i + 1] == t_int) {
				paper_id[i] = t_int;
				break;
			}
			if (paper_id[i+1] < t_int)
			{
				paper_id[i] = paper_id[i + 1];
			}
		}
		*/
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return getTimeElapsed(start, stop);

}

double CuSource::print_table(const char * file_name)
{
	clock_t c_start, c_end;

	c_start = startTimer();

	FILE * p_file;
	std::string sorted_file_name(file_name);
	int i_value;
	std::string s_value;

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_sm_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");
	
	for (size_t i = 0; i < rows; i++)
	{

		switch (column_decide % 3)
		{
		case 0:
			s_value = sorted_col_string[i];
			printf_stream(p_file, "%s\n", s_value.c_str());
			break;
		case 1:
			printf_stream(p_file, "%d\n", paper_id[i]);
			break;
		case 2:
			s_value = sorted_col_string[i];
			printf_stream(p_file, "%s\n", s_value.c_str());
			break;
		}

	}

	fclose(p_file);

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end);

}

void CuSource::initializeTimer(hipEvent_t * start, hipEvent_t * end) 
{
	hipEventCreate(start);
	hipEventCreate(end);
}

void CuSource::timerEventRecord(hipEvent_t * timer)
{
	hipEventRecord(*timer);
}

void CuSource::timerEventSync(hipEvent_t * timer)
{
	hipEventSynchronize(*timer);
}

double CuSource::getTimeElapsed(hipEvent_t start, hipEvent_t end)
{
	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, end);

	return milliseconds/1000;
}

clock_t CuSource::startTimer()
{
	return clock();
}

clock_t CuSource::endTimer()
{
	return clock();
}

double CuSource::getTimeElapsed(clock_t start, clock_t end)
{
	return (end - start) / static_cast<double>(CLOCKS_PER_SEC);
}

