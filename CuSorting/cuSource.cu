#include "hip/hip_runtime.h"

#include "cuSource.h"
#include <hip/hip_runtime.h>

#include "time.h"

#include "GPU_Sorting_Functions.cuh"

double CuSource::sort() {
	
	clock_t c_start, c_end;
	c_start = startTimer();

	int j, k;

	unsigned int num_blocks;
	unsigned int num_threads_per_block;
	bool sorted;
	int pre_val;
	size_t arr_size, num_arr;
	size_t left, right;
	size_t gpu_loop;

	switch (column_decide % 3)
	{
	case 0:
		break;
	case 1:
		// call sorting kernel with
		// d_paperIdWrapper
		switch ((column_decide - 1) / 3)
		{
		case 0:
			gpuErrchk(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));
			left = 0;
			right = static_cast<size_t>(rows - 1);
			quicksort_int<<< 1, 1>>>(d_int, left, right, 0);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
			break;
		case 1:
			for (k = 2; k <= NUM_VALS; k <<= 1) {
				for (j = k >> 1; j>0; j = j >> 1) {
					bitonic_sort_int <<<BLOCKS, THREADS >>>(d_int, j, k);
					gpuErrchk(hipPeekAtLastError());
				}
			}
			break;
		case 2:
			gpu_loop = 1024;
			num_blocks = static_cast<unsigned int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_int<<<num_blocks, WID_BLOCK>>>(d_int, rows);
				gpuErrchk(hipPeekAtLastError());
			}
			break;
		default:
			break;
		}
		break;
	case 2:
		break;
	}

	c_end = endTimer();
	return getTimeElapsed(c_start, c_end);

}

double CuSource::MemAllo(const char* file_name)
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	c_start = startTimer();

	Source::MemAllo(file_name);

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		if (((column_decide - 1) / 3) == 1)
		{
			gpuErrchk(
				hipMalloc((void **)&d_int, NUM_VALS * sizeof(TYPE_PAPER_ID))
			);
		}
		else {
			gpuErrchk(
				hipMalloc((void **)&d_int, rows * sizeof(TYPE_PAPER_ID))
			);
		}
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end)+getTimeElapsed(start, stop);
}

double CuSource::preSorting()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	size_t i;

	c_start = startTimer();

	Source::preSorting();

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		// NOT READY
		//	hipMalloc((void **)d_institution_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_institution_name = new char[length_institution_name + 1];
		// TODO: hipMemcpy
		break;
	case 1:
		if (((column_decide - 1) / 3) == 1)
			bitonic_sort_int_initMax<<<BLOCKS, THREADS>>>(d_int);
		gpuErrchk(
			hipMemcpy(d_int, paper_id,
				rows * sizeof(int),
				hipMemcpyHostToDevice
			)
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)d_subject_name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}


double CuSource::MemFree()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipFree(d_int)
		);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	c_start = startTimer();

	Source::MemFree();

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}



double CuSource::postSorting()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	int t_int;
	size_t i;

	switch (column_decide % 3)
	{
	case 0:
		//	hipFree(d_institution_name);
		//	delete[](h_institution_name);
		break;
	case 1:
		gpuErrchk(
			hipMemcpy(paper_id, d_int,
				rows * sizeof(int),
				hipMemcpyDeviceToHost
			)
		);
		break;
	case 2:
		//	hipFree(d_subject_name);
		//	delete[](h_subject_name);
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return getTimeElapsed(start, stop);

}

double CuSource::print_table(const char * file_name)
{
	clock_t c_start, c_end;

	c_start = startTimer();

	FILE * p_file;
	std::string sorted_file_name(file_name);
	int i_value;
	std::string s_value;

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_sm_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");
	
	for (size_t i = 0; i < rows; i++)
	{

		switch (column_decide % 3)
		{
		case 0:
			s_value = sorted_col_string[i];
			printf_stream(p_file, "%s\n", s_value.c_str());
			break;
		case 1:
			printf_stream(p_file, "%d\n", paper_id[i]);
			break;
		case 2:
			s_value = sorted_col_string[i];
			printf_stream(p_file, "%s\n", s_value.c_str());
			break;
		}

	}

	fclose(p_file);

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end);

}

void CuSource::initializeTimer(hipEvent_t * start, hipEvent_t * end) 
{
	hipEventCreate(start);
	hipEventCreate(end);
}

void CuSource::timerEventRecord(hipEvent_t * timer)
{
	hipEventRecord(*timer);
}

void CuSource::timerEventSync(hipEvent_t * timer)
{
	hipEventSynchronize(*timer);
}

double CuSource::getTimeElapsed(hipEvent_t start, hipEvent_t end)
{
	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, end);

	return milliseconds/1000;
}

clock_t CuSource::startTimer()
{
	return clock();
}

clock_t CuSource::endTimer()
{
	return clock();
}

double CuSource::getTimeElapsed(clock_t start, clock_t end)
{
	return (end - start) / static_cast<double>(CLOCKS_PER_SEC);
}

