
#include "cuBiggerSource.h"
#include <hip/hip_runtime.h>

#include "GPU_Sorting_Functions.cuh"

#define NUM_BLOCK 65535
#define WID_BLOCK 1024

void CuBiggerSource::sort() {
	switch (column_decide % 3)
	{
	case 0:
		//call sorting kernel with
		//d_paperIdWrapper
		switch (column_decide / 3)
		{
		case 2:
			odd_even_sort_results_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
		default:
			break;
		}
		break;
	case 1:

		break;
	case 2:
		//call sorting kernel with
		//d_rollNumberWrapper
		break;
	default:
		break;
	}
}

void CuBiggerSource::MemAllo(const char* file_name)
{
	BiggerSource::MemAllo(file_name);

	switch (column_decide % 3)
	{
	case 0:
		paperIdWrapper = (struct PaperIdWrapper_Results *) malloc(rows * sizeof(struct PaperIdWrapper_Results));
		for (size_t i = 0; i < rows; i++) {
			paperIdWrapper[i].paper_id = paper_id[i];
			paperIdWrapper[i].classPtr = &resultsDataStructure[i];
		}
		hipMalloc((void **)d_paperIdWrapper, rows * sizeof(struct PaperIdWrapper_Results));
		hipMemcpy(d_paperIdWrapper, paperIdWrapper,
			rows * sizeof(struct PaperIdWrapper_Results),
			hipMemcpyHostToDevice
		);
		break;
	case 1:
		// TODO: 
		break;
	case 2:
		rollNumberWrapper = (struct RollNumberWrapper_Results *) malloc(rows * sizeof(struct RollNumberWrapper_Results));
		for (size_t i = 0; i < rows; i++) {
			rollNumberWrapper[i].rollnumber = rollnumber[i];
			rollNumberWrapper[i].classPtr = &resultsDataStructure[i];
		}
		hipMalloc((void **)d_rollNumberWrapper, rows * sizeof(struct PaperIdWrapper_Results));
		hipMemcpy(d_rollNumberWrapper, rollNumberWrapper,
			rows * sizeof(struct PaperIdWrapper_Results),
			hipMemcpyHostToDevice
		);
		break;

	}

}

void CuBiggerSource::MemFree() {

	switch (column_decide % 3)
	{
	case 0:
		hipFree(d_paperIdWrapper);
		free(paperIdWrapper);
		break;
	case 1:
		break;
	case 2:
		hipFree(d_rollNumberWrapper);
		free(rollNumberWrapper);
		break;
	default:
		break;
	}


	BiggerSource::MemFree();

}


void CuBiggerSource::postSorting()
{
	switch (column_decide % 3)
	{
	case 0:
		hipMemcpy(paperIdWrapper, d_paperIdWrapper,
			rows * sizeof(struct PaperIdWrapper_Results),
			hipMemcpyDeviceToHost
		);
		for (size_t i = 0; i < rows; i++) {
			resultsDataStructure[i] = *paperIdWrapper[i].classPtr;
		}
		break;
	case 1:
		// TODO: 
		break;
	case 2:
		hipMemcpy(rollNumberWrapper, d_rollNumberWrapper,
			rows * sizeof(struct PaperIdWrapper_Results),
			hipMemcpyDeviceToHost
		);
		for (size_t i = 0; i < rows; i++) {
			resultsDataStructure[i] = *rollNumberWrapper[i].classPtr;
		}
		break;
	}

}

void CuBiggerSource::write_file(const char * file_name, ResultsDataStructure * resultsDataStructure)
{
	FILE * p_file;
	std::string sorted_file_name(file_name);

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_big_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");

	std::vector<std::string>::iterator iter;
	for (iter = headers.begin();
		iter != headers.end(); ++iter) {
		if (iter != headers.begin())
			printf_stream(p_file, ",");
		printf_stream(p_file, "%s", (*iter).c_str());
	}

	printf_stream(p_file, "\n");

	struct ResultsDSHolder resultsDSHolder;

	for (size_t i = 0; i < rows; i++) {

		resultsDataStructure[i].getValue(&resultsDSHolder);

		printf_stream(p_file, "%d,%s,%s,%s,%s,%d,%s,%d,%s,%lld,%s,%lld,%lld,%d,%s,%s,%s,%s\n",
			resultsDSHolder.scheme_prog_code, resultsDSHolder.prepared_date.c_str(),
			resultsDSHolder.declared_date.c_str(), resultsDSHolder.prog_name.c_str(),
			resultsDSHolder.prog_sem_year.c_str(), resultsDSHolder.batch,
			resultsDSHolder.examination.c_str(), resultsDSHolder.institution_code,
			resultsDSHolder.institution_name.c_str(), resultsDSHolder.rollnumber,
			resultsDSHolder.name.c_str(), resultsDSHolder.sid, resultsDSHolder.result_scheme_id,
			resultsDSHolder.paper_id, resultsDSHolder.credits.c_str(), resultsDSHolder.minor.c_str(),
			resultsDSHolder.major.c_str(), resultsDSHolder.total.c_str());

	}

	fclose(p_file);
}
