
#include "cuBiggerSource.h"
#include <hip/hip_runtime.h>

#include "GPU_Sorting_Functions.cuh"

#define NUM_BLOCK 65535
#define WID_BLOCK 1024

void CuBiggerSource::sort() {

	unsigned int num_blocks;

	switch (column_decide % 3)
	{
	case 0:
		//call sorting kernel with
		//d_rollNumberWrapper
		switch ((column_decide - 1) / 3)
		{
		case 0:
			//quicksort_results_rollnumber<<<NUM_BLOCK, WID_BLOCK>>>(d_rollNumberWrapper);
			//hipDeviceSynchronize();
			break;
		case 1:
			//shellsort_results_rollnumber<<<NUM_BLOCK, WID_BLOCK>>>(d_rollNumberWrapper);
			//hipDeviceSynchronize();
			break;
		case 2:
			num_blocks = static_cast<int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_llong<<<num_blocks, WID_BLOCK>>>(d_llong, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}
			break;
		}
		break;
	case 1:
		//call sorting kernel with
		//d_paperIdWrapper
		switch ((column_decide-1) / 3)
		{
		case 0:
			//quicksort_results_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			//hipDeviceSynchronize();
			break;
		case 1:
			//shellsort_results_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			//hipDeviceSynchronize();
			break;
		case 2:
			num_blocks = static_cast<int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_int<<<num_blocks, WID_BLOCK>>>(d_int, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}
			break;
		default:
			break;
		}
		break;
	case 2:

		break;
	default:
		break;
	}
}


void CuBiggerSource::MemAllo(const char* file_name)
{
	BiggerSource::MemAllo(file_name);

	switch (column_decide % 3)
	{
	case 0:
		gpuErrchk(
			hipMalloc((void **)&d_llong, rows * sizeof(long long))
		);
		break;
	case 1:
		gpuErrchk(
			hipMalloc((void **)&d_int, rows * sizeof(int))
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}


}


void CuBiggerSource::preSorting()
{
	BiggerSource::preSorting();

	switch (column_decide % 3)
	{
	case 0:
		hipMemcpy(d_llong, rollnumber,
			rows * sizeof(long long),
			hipMemcpyHostToDevice
		);
		break;
	case 1:
		hipMemcpy(d_int, paper_id,
			rows * sizeof(int),
			hipMemcpyHostToDevice
		);
		break;
	case 2:
		// TODO: 
		break;
	}

}

void CuBiggerSource::MemFree() {

	switch (column_decide % 3)
	{
	case 0:
		hipFree(d_llong);
		break;
	case 1:
		hipFree(d_int);
		break;
	case 2:
		break;
	default:
		break;
	}

	BiggerSource::MemFree();

}


void CuBiggerSource::postSorting()
{
	switch (column_decide % 3)
	{
	case 0:
		hipMemcpy(rollnumber, d_llong,
			rows * sizeof(long long),
			hipMemcpyDeviceToHost
		);
		break;
	case 1:
		hipMemcpy(paper_id, d_int,
			rows * sizeof(int),
			hipMemcpyDeviceToHost
		);
		break;
	case 2:
		// TODO: 
		break;
	}

}

void CuBiggerSource::print_table(const char * file_name)
{
	FILE * p_file;
	std::string sorted_file_name(file_name);

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_big_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");

	for (size_t i = 0; i < rows; i++) {

		switch ((column_decide-1) % 3)
		{
		case 0:
			printf_stream(p_file, "%lld\n", rollnumber[i]);
			break;
		case 1:
			printf_stream(p_file, "%d\n", paper_id[i]);
			break;
		case 2:
			printf_stream(p_file, "%s\n", name[i].c_str());
			break;
		}

	}

	fclose(p_file);
}
