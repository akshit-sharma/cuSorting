#include "hip/hip_runtime.h"

#include "cuBiggerSource.h"
#include <hip/hip_runtime.h>

#include "time.h"

#include "GPU_Sorting_Functions.cuh"

#define NUM_BLOCK 65535
#define WID_BLOCK 1024

double CuBiggerSource::sort() {

	hipEvent_t start, stop;
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	unsigned int num_blocks;
	unsigned int num_threads_per_block;
	size_t arr_size, num_arr;
	size_t left, right;
	float progress, last_prog;

	last_prog = 0.0f;

	switch (column_decide % 3)
	{
	case 0:
		//call sorting kernel with
		//d_rollNumberWrapper
		switch ((column_decide - 1) / 3)
		{
		case 0:
			left = 0;
			right = static_cast<size_t>(rows - 1);
			quicksort_llong<<< 1, 1 >>>(d_llong, left, right, 0);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
			break;
		case 1:
			for (unsigned int i = rows / 2; i > 0; i /= 2) {
				num_threads_per_block = rows;
				num_arr = i + 1;
				if ((num_threads_per_block - 1) / WID_BLOCK > 0) {
					num_blocks = (num_threads_per_block / WID_BLOCK) + 1;
					num_threads_per_block = WID_BLOCK;
				}
				arr_size = rows / i;

				//transfer data from d_int to d_xtra_int
				shellsort_llong_front << <num_blocks, num_threads_per_block >> >(d_llong, rows, num_arr - 1, arr_size, d_xtra_llong);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());

				num_blocks = static_cast<unsigned int>(rows / WID_BLOCK) + 1;
				num_blocks = (num_blocks / 2) + 1;
				for (unsigned int i = 0; i < (arr_size / 2) + 1; i++) {
					odd_even_sort_llong_xtra << <num_blocks, WID_BLOCK >> >(d_xtra_llong, rows, num_arr, arr_size);
					gpuErrchk(hipPeekAtLastError());
					gpuErrchk(hipDeviceSynchronize());
				}

				num_threads_per_block = rows;
				if ((num_threads_per_block - 1) / WID_BLOCK > 0) {
					num_blocks = (num_threads_per_block / WID_BLOCK) + 1;
					num_threads_per_block = WID_BLOCK;
				}
				arr_size = rows / i;

				//transfer data from d_int to d_xtra_int
				if (i == 1) {
					gpuErrchk(
						hipMemcpy(d_llong, d_xtra_llong,
							rows * sizeof(long long),
							hipMemcpyDeviceToDevice
						)
					);
				}
				else {
					shellsort_llong_back << <num_blocks, num_threads_per_block >> > (d_llong, rows, num_arr - 1, arr_size, d_xtra_llong);
					gpuErrchk(hipPeekAtLastError());
					gpuErrchk(hipDeviceSynchronize());
				}
			}
			break;
		case 2:
			num_blocks = static_cast<unsigned int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			//printf_stream(stdout,"\n");
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_llong<<<num_blocks, WID_BLOCK>>>(d_llong, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
				/*
				progress = (static_cast<float>(i) * 100) / static_cast<float>(rows);
				if (progress >(static_cast<int>(last_prog + 10) % 300)) {
					printf_stream(stdout, "\rDone with %7.4f percent \n",
						progress);
					last_prog = progress;
				}
				*/
			}
			break;
		}
		break;
	case 1:
		//call sorting kernel with
		//d_paperIdWrapper
		switch ((column_decide-1) / 3)
		{
		case 0:
			gpuErrchk(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));
			left = 0;
			right = static_cast<int>(rows - 1);
			quicksort_int<<< 1, 1>>>(d_int, left, right, 0);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
			break;
		case 1:

			for (unsigned int i = rows / 2; i > 0; i /= 2) {
				num_threads_per_block = rows;
				num_arr = i + 1;
				if ((num_threads_per_block - 1) / WID_BLOCK > 0) {
					num_blocks = (num_threads_per_block / WID_BLOCK) + 1;
					num_threads_per_block = WID_BLOCK;
				}
				arr_size = rows / i;

				//transfer data from d_int to d_xtra_int
				shellsort_int << <num_blocks, num_threads_per_block >> >(d_int, rows, num_arr - 1, arr_size, d_xtra_int);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());

				num_blocks = static_cast<unsigned int>(rows / WID_BLOCK) + 1;
				num_blocks = (num_blocks / 2) + 1;
				if (arr_size > UINT_MAX)
					printf_s("NOT GOOD");
				for (unsigned int i = 0; i < (arr_size / 2) + 1; i++) {
					odd_even_sort_int_xtra << <num_blocks, WID_BLOCK >> >(d_xtra_int, rows, num_arr, arr_size);
					gpuErrchk(hipPeekAtLastError());
					gpuErrchk(hipDeviceSynchronize());
				}

				num_threads_per_block = rows;
				if ((num_threads_per_block - 1) / WID_BLOCK > 0) {
					num_blocks = (num_threads_per_block / WID_BLOCK) + 1;
					num_threads_per_block = WID_BLOCK;
				}
				arr_size = rows / i;

				//transfer data from d_int to d_xtra_int
				if (i == 1) {
					gpuErrchk(
						hipMemcpy(d_int, d_xtra_int,
							rows * sizeof(int),
							hipMemcpyDeviceToDevice
						)
					);
				}
				else {
					shellsort_int_back << <num_blocks, num_threads_per_block >> > (d_int, rows, num_arr - 1, arr_size, d_xtra_int);
					gpuErrchk(hipPeekAtLastError());
					gpuErrchk(hipDeviceSynchronize());
				}
			}
			break;
		case 2:
			num_blocks = static_cast<int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			//printf_stream(stdout, "\n");
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_int<<<num_blocks, WID_BLOCK>>>(d_int, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
				/*
				progress = (static_cast<float>(i) * 100) / static_cast<float>(rows);
				if (progress > (static_cast<int>(last_prog+10)%300)) {
					printf_stream(stdout, "\rDone with %8.4f percent of result of paper_id\n",
						progress);
					last_prog = progress;
				}
				*/
			}
			break;
		default:
			break;
		}
		break;
	case 2:

		break;
	default:
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return getTimeElapsed(start, stop);

}


double CuBiggerSource::MemAllo(const char* file_name)
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	c_start = startTimer();

	BiggerSource::MemAllo(file_name);

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		gpuErrchk(
			hipMalloc((void **)&d_llong, rows * sizeof(long long))
		);
		gpuErrchk(
			hipMalloc((void **)&d_xtra_llong, rows * sizeof(long long))
		);
		break;
	case 1:
		gpuErrchk(
			hipMalloc((void **)&d_int, rows * sizeof(int))
		);
		gpuErrchk(
			hipMalloc((void **)&d_xtra_int, rows * sizeof(int))
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}


double CuBiggerSource::preSorting()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	c_start = startTimer();

	BiggerSource::preSorting();

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		hipMemcpy(d_llong, rollnumber,
			rows * sizeof(long long),
			hipMemcpyHostToDevice
		);
		break;
	case 1:
		hipMemcpy(d_int, paper_id,
			rows * sizeof(int),
			hipMemcpyHostToDevice
		);
		break;
	case 2:
		// TODO: 
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}

double CuBiggerSource::MemFree() {

	clock_t c_start, c_end;
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		hipFree(d_xtra_llong);
		hipFree(d_llong);
		break;
	case 1:
		hipFree(d_xtra_int);
		hipFree(d_int);
		break;
	case 2:
		break;
	default:
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	c_start = startTimer();

	BiggerSource::MemFree();

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}


double CuBiggerSource::postSorting()
{
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		hipMemcpy(rollnumber, d_llong,
			rows * sizeof(long long),
			hipMemcpyDeviceToHost
		);
		break;
	case 1:
		hipMemcpy(paper_id, d_int,
			rows * sizeof(int),
			hipMemcpyDeviceToHost
		);
		break;
	case 2:
		// TODO: 
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return getTimeElapsed(start, stop);

}

double CuBiggerSource::print_table(const char * file_name)
{
	clock_t c_start, c_end;

	c_start = startTimer();

	FILE * p_file;
	std::string sorted_file_name(file_name);

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_big_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");

	for (size_t i = 0; i < rows; i++) {

		switch ((column_decide-1) % 3)
		{
		case 0:
			printf_stream(p_file, "%lld\n", rollnumber[i]);
			break;
		case 1:
			printf_stream(p_file, "%d\n", paper_id[i]);
			break;
		case 2:
			printf_stream(p_file, "%s\n", name[i].c_str());
			break;
		}

	}

	fclose(p_file);

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end);
}

void CuBiggerSource::initializeTimer(hipEvent_t * start, hipEvent_t * end)
{
	hipEventCreate(start);
	hipEventCreate(end);
}

void CuBiggerSource::timerEventRecord(hipEvent_t * timer)
{
	hipEventRecord(*timer);
}

void CuBiggerSource::timerEventSync(hipEvent_t * timer)
{
	hipEventSynchronize(*timer);
}

double CuBiggerSource::getTimeElapsed(hipEvent_t start, hipEvent_t end)
{
	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, end);

	return milliseconds / 1000;
}

clock_t CuBiggerSource::startTimer()
{
	return clock();
}

clock_t CuBiggerSource::endTimer()
{
	return clock();
}

double CuBiggerSource::getTimeElapsed(clock_t start, clock_t end)
{
	return (end - start) / static_cast<double>(CLOCKS_PER_SEC);
}

