
#include "cuBiggerSource.h"
#include <hip/hip_runtime.h>

#include "GPU_Sorting_Functions.cuh"

#define NUM_BLOCK 65535
#define WID_BLOCK 1024

void CuBiggerSource::sort(int column) {
	switch (column % 3)
	{
	case 0:
		//call sorting kernel with
		//d_paperIdWrapper
		switch (column / 3)
		{
		case 2:
			odd_even_sort_results_paperid<<<NUM_BLOCK, WID_BLOCK>>>(d_paperIdWrapper);
			hipDeviceSynchronize();
		default:
			break;
		}
		break;
	case 1:

		break;
	case 2:
		//call sorting kernel with
		//d_rollNumberWrapper
		break;
	default:
		break;
	}
}

void CuBiggerSource::MemAllo(const char* file_name)
{
	CuBiggerSource::MemAllo(file_name);

	switch (column % 3)
	{
	case 0:
		paperIdWrapper = (struct PaperIdWrapper_Results *) malloc(rows * sizeof(struct PaperIdWrapper_Results));
		for (size_t i = 0; i < rows; i++) {
			paperIdWrapper[i].paper_id = paper_id[i];
			paperIdWrapper[i].classPtr = &resultsDataStructure[i];
		}
		hipMalloc((void **)d_paperIdWrapper, rows * sizeof(struct PaperIdWrapper_Results));
		hipMemcpy(d_paperIdWrapper, paperIdWrapper,
			rows * sizeof(struct PaperIdWrapper_Results),
			hipMemcpyHostToDevice
		);
		break;
	case 1:
		// TODO: 
		break;
	case 2:
		rollNumberWrapper = (struct RollNumberWrapper_Results *) malloc(rows * sizeof(struct RollNumberWrapper_Results));
		for (size_t i = 0; i < rows; i++) {
			rollNumberWrapper[i].rollnumber = rollnumber[i];
			rollNumberWrapper[i].classPtr = &resultsDataStructure[i];
		}
		hipMalloc((void **)d_rollNumberWrapper, rows * sizeof(struct PaperIdWrapper_Results));
		hipMemcpy(d_rollNumberWrapper, rollNumberWrapper,
			rows * sizeof(struct PaperIdWrapper_Results),
			hipMemcpyHostToDevice
		);
		break;

	}

}

void CuBiggerSource::MemFree() {

	switch (column % 3)
	{
	case 0:
		hipFree(d_paperIdWrapper);
		free(paperIdWrapper);
		break;
	case 1:
		break;
	case 2:
		hipFree(d_rollNumberWrapper);
		free(rollNumberWrapper);
		break;
	default:
		break;
	}


	BiggerSource::MemFree();

}


