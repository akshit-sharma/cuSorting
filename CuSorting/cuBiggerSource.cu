#include "hip/hip_runtime.h"

#include "cuBiggerSource.h"
#include <hip/hip_runtime.h>

#include "time.h"

#include "GPU_Sorting_Functions.cuh"

#define NUM_BLOCK 65535
#define WID_BLOCK 1024

double CuBiggerSource::sort() {

	hipEvent_t start, stop;
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	int j, k;

	unsigned int num_blocks;
	unsigned int num_threads_per_block;
	size_t arr_size, num_arr;
	size_t left, right;
	float progress, last_prog;
	int h_ans;

	const int zero = 0;

	last_prog = 0.0f;

	switch (column_decide % 3)
	{
	case 0:
		//call sorting kernel with
		//d_rollNumberWrapper
		switch ((column_decide - 1) / 3)
		{
		case 0:
			left = 0;
			right = static_cast<size_t>(rows - 1);
			quicksort_llong<<< 1, 1 >>>(d_llong, left, right, 0);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
			break;
		case 1:
			num_blocks = max2Rows / THREADS + 1;
			for (k = 2; k <= max2Rows; k <<= 1) {
				for (j = k >> 1; j>0; j = j >> 1) {
					bitonic_sort_llong << <num_blocks, THREADS >> >(d_llong, j, k);
					gpuErrchk(hipPeekAtLastError());
				}
			}
			break;
		case 2:
			num_blocks = static_cast<unsigned int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_llong<<<num_blocks, WID_BLOCK>>>(d_llong, rows);
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
			}
			break;
		}
		break;
	case 1:
		//call sorting kernel with
		//d_paperIdWrapper
		switch ((column_decide-1) / 3)
		{
		case 0:
			gpuErrchk(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));
			left = 0;
			right = static_cast<int>(rows - 1);
			quicksort_int<<< 1, 1>>>(d_int, left, right, 0);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
			break;
		case 1:
			num_blocks = max2Rows / THREADS + 1;
			for (k = 2; k <= max2Rows; k <<= 1) {
				for (j = k >> 1; j>0; j = j >> 1) {
					bitonic_sort_int << <num_blocks, THREADS >> >(d_int, j, k);
					gpuErrchk(hipPeekAtLastError());
				}
			}
			break;
		case 2:
			num_blocks = static_cast<int>(rows / WID_BLOCK) + 1;
			num_blocks = (num_blocks / 2) + 1;
			for (unsigned int i = 0; i < rows; i++) {
				odd_even_sort_int<<<num_blocks, WID_BLOCK>>>(d_int, rows);
				gpuErrchk(hipPeekAtLastError());
			}
			break;
		default:
			break;
		}
		break;
	case 2:

		break;
	default:
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return getTimeElapsed(start, stop);
	
}


double CuBiggerSource::MemAllo(const char* file_name)
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	c_start = startTimer();

	BiggerSource::MemAllo(file_name);

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		gpuErrchk(
			hipMalloc((void **)&d_llong, NUM_VALS * sizeof(long long))
		);
		break;
	case 1:
		gpuErrchk(
			hipMalloc((void **)&d_int, NUM_VALS * sizeof(int))
		);
		break;
	case 2:
		// NOT READY
		//	hipMalloc((void **)name, rows * sizeof(char));
		// TODO: fig. out a way for string to gpu
		//	h_subject_name = new char[length_subject_name + 1];
		// TODO: hipMemcpy
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}


double CuBiggerSource::preSorting()
{
	clock_t c_start, c_end;
	hipEvent_t start, stop;
	c_start = startTimer();

	BiggerSource::preSorting();

	c_end = endTimer();
	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		if (((column_decide - 1) / 3) == 1)
			bitonic_sort_llong_initMax <<<BLOCKS, THREADS >>>(d_llong, rows);
		gpuErrchk(
			hipMemcpy(d_llong, rollnumber,
				rows * sizeof(long long),
				hipMemcpyHostToDevice
			)
		);
		break;
	case 1:
		if (((column_decide - 1) / 3) == 1)
			bitonic_sort_int_initMax << <BLOCKS, THREADS >> >(d_int, rows);
		gpuErrchk(
			hipMemcpy(d_int, paper_id,
				rows * sizeof(int),
				hipMemcpyHostToDevice
			)
		);
		break;
	case 2:
		// TODO: 
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}

double CuBiggerSource::MemFree() {

	clock_t c_start, c_end;
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		gpuErrchk(
			hipFree(d_llong)
		);
		break;
	case 1:
		gpuErrchk(
			hipFree(d_int)
		);
		break;
	case 2:
		break;
	default:
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	c_start = startTimer();

	BiggerSource::MemFree();

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end) + getTimeElapsed(start, stop);

}


double CuBiggerSource::postSorting()
{
	hipEvent_t start, stop;

	initializeTimer(&start, &stop);
	timerEventRecord(&start);

	switch (column_decide % 3)
	{
	case 0:
		hipMemcpy(rollnumber, d_llong,
			rows * sizeof(long long),
			hipMemcpyDeviceToHost
		);
		break;
	case 1:
		hipMemcpy(paper_id, d_int,
			rows * sizeof(int),
			hipMemcpyDeviceToHost
		);
		break;
	case 2:
		// TODO: 
		break;
	}

	timerEventRecord(&stop);
	timerEventSync(&stop);

	return getTimeElapsed(start, stop);

}

double CuBiggerSource::print_table(const char * file_name)
{
	clock_t c_start, c_end;

	c_start = startTimer();

	FILE * p_file;
	std::string sorted_file_name(file_name);

	sorted_file_name.replace(sorted_file_name.end() - 4,
		sorted_file_name.end(), "_big_gpu_");

	sorted_file_name += std::to_string(init_num);
	sorted_file_name += ".csv";

	fopen_stream(&p_file, sorted_file_name.c_str(), "w");

	for (size_t i = 0; i < rows; i++) {

		switch ((column_decide-1) % 3)
		{
		case 0:
			printf_stream(p_file, "%lld\n", rollnumber[i]);
			break;
		case 1:
			printf_stream(p_file, "%d\n", paper_id[i]);
			break;
		case 2:
			printf_stream(p_file, "%s\n", name[i].c_str());
			break;
		}

	}

	fclose(p_file);

	c_end = endTimer();

	return  getTimeElapsed(c_start, c_end);
}

void CuBiggerSource::initializeTimer(hipEvent_t * start, hipEvent_t * end)
{
	hipEventCreate(start);
	hipEventCreate(end);
}

void CuBiggerSource::timerEventRecord(hipEvent_t * timer)
{
	hipEventRecord(*timer);
}

void CuBiggerSource::timerEventSync(hipEvent_t * timer)
{
	hipEventSynchronize(*timer);
}

double CuBiggerSource::getTimeElapsed(hipEvent_t start, hipEvent_t end)
{
	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, end);

	return milliseconds / 1000;
}

clock_t CuBiggerSource::startTimer()
{
	return clock();
}

clock_t CuBiggerSource::endTimer()
{
	return clock();
}

double CuBiggerSource::getTimeElapsed(clock_t start, clock_t end)
{
	return (end - start) / static_cast<double>(CLOCKS_PER_SEC);
}

