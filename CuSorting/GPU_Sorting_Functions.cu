#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"
#include "cuSource.h"

__global__ void checkArray(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme, size_t maxLimit)
{
	size_t threadIndex;

	threadIndex = threadIdx.x + blockIdx.x*blockDim.x;

	if (threadIndex + 1 < maxLimit) {
		if (d_PaperIdWrapper_Scheme[threadIndex].paper_id > d_PaperIdWrapper_Scheme[threadIndex+1].paper_id)
		{
			printf("Array not sorted \n %d is before %d \n\n",
				d_PaperIdWrapper_Scheme[threadIndex].paper_id,
				d_PaperIdWrapper_Scheme[threadIndex + 1].paper_id);
		}
	}

}

__device__ __host__ void printPaperIdCombinationScheme(PaperIdWrapper_Scheme * single_PaperIdWrapper, size_t threadIndex) {
	int paperId;
	SchemeDataStructure * schemeDataStructure;

	paperId = single_PaperIdWrapper->paper_id;
	schemeDataStructure = single_PaperIdWrapper->classPtr;

	/*
	printf("paper id %d \n classPtr is %llu threadIdx is %llu \n",
		single_PaperIdWrapper->paper_id,
		single_PaperIdWrapper->classPtr,
		threadIndex
	);
	*/

}


__global__ void odd_even_sort_scheme_paperid_swap_edges_only(PaperIdWrapper_Scheme * d_PaperIdWrapper, size_t maxLimit)
{
	size_t threadIndex;
	size_t arrayIndex;
	PaperIdWrapper_Scheme temp;
	size_t internal_max;

	threadIndex = (threadIdx.x + 1) * blockDim.x;

	arrayIndex = threadIndex * 2;

	//printf("kernel called \n");

	if (arrayIndex + 1 < maxLimit) {
		if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
		{
			//			printf("\nbefore even swap from gpu \n");
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex+1], threadIndex);
//			atomicExch(&d_PaperIdWrapper[arrayIndex + 1].paper_id, atomicExch(&d_PaperIdWrapper[arrayIndex].paper_id, d_PaperIdWrapper[arrayIndex + 1].paper_id));
//			atomicExch(&d_PaperIdWrapper[arrayIndex + 1].classPtr, atomicExch(&d_PaperIdWrapper[arrayIndex].classPtr, d_PaperIdWrapper[arrayIndex + 1].classPtr));
			temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
//			temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
			d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
//			d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
			d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
//			d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
			//			printf("\nafter even swap from gpu \n");
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
			{
				//				printf("before odd swap \n");
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
//				atomicExch(&d_PaperIdWrapper[arrayIndex + 1].paper_id, atomicExch(&d_PaperIdWrapper[arrayIndex].paper_id, d_PaperIdWrapper[arrayIndex + 1].paper_id));
//				atomicExch(&d_PaperIdWrapper[arrayIndex + 1].classPtr, atomicExch(&d_PaperIdWrapper[arrayIndex].classPtr, d_PaperIdWrapper[arrayIndex + 1].classPtr));
				temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
//				temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
				d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
//				d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
				d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
//				d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
				//				printf("after odd swap \n");
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
			}
		}
		__syncthreads();
	}

}


__global__ void odd_even_sort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper, size_t maxLimit, int offset)
{
	size_t threadIndex;
	size_t arrayIndex;
	size_t internal_max;
	PaperIdWrapper_Scheme temp;

	threadIndex = threadIdx.x + blockIdx.x*blockDim.x;
	threadIndex += offset;

	internal_max = (blockIdx.x + 1) * blockDim.x;
	internal_max += offset;

	arrayIndex = threadIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
		{
			//			printf("SWAP 1 called by %d on %d & %d \n", threadIndex, arrayIndex, arrayIndex + 1);
			SWAP_FOR_GPU_SPEC(d_PaperIdWrapper, paper_id, temp.paper_id, arrayIndex, arrayIndex + 1);
			SWAP_FOR_GPU_SPEC(d_PaperIdWrapper, classPtr, temp.classPtr, arrayIndex, arrayIndex + 1);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
			{
				//			printf("SWAP 2 called by %d on %d & %d \n", threadIndex, arrayIndex, arrayIndex + 1);
				SWAP_FOR_GPU_SPEC(d_PaperIdWrapper, paper_id, temp.paper_id, arrayIndex, arrayIndex + 1);
				SWAP_FOR_GPU_SPEC(d_PaperIdWrapper, classPtr, temp.classPtr, arrayIndex, arrayIndex + 1);
			}
		}
		__syncthreads();
	}
}

__global__ void odd_even_sort_scheme_paperid_separate(int * d_paper_id, SchemeDataStructure ** d_classPtr, size_t maxLimit, int offset) {
	size_t threadIndex;
	size_t arrayIndex;
	size_t internal_max;
	int t_int;
	SchemeDataStructure * t_classPtr;

	threadIndex = threadIdx.x + blockIdx.x*blockDim.x;
	threadIndex += offset;

	internal_max = (blockIdx.x + 1) * blockDim.x;
	internal_max += offset;

	arrayIndex = threadIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_paper_id[arrayIndex] > d_paper_id[arrayIndex + 1])
		{
			//			printf("SWAP 1 called by %d on %d & %d \n", threadIndex, arrayIndex, arrayIndex + 1);
			SWAP_FOR_GPU(d_paper_id, t_int, arrayIndex, arrayIndex + 1);
			SWAP_FOR_GPU(d_classPtr, t_classPtr, arrayIndex, arrayIndex + 1);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_paper_id[arrayIndex] > d_paper_id[arrayIndex + 1])
			{
				//			printf("SWAP 2 called by %d on %d & %d \n", threadIndex, arrayIndex, arrayIndex + 1);
				SWAP_FOR_GPU(d_paper_id, t_int, arrayIndex, arrayIndex + 1);
				SWAP_FOR_GPU(d_classPtr, t_classPtr, arrayIndex, arrayIndex + 1);
			}
		}
		__syncthreads();
	}
}

__global__ void odd_even_sort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper)
{
	// Function body

}

__global__ void odd_even_sort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

__global__ void shellsort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme)
{
	// Function body

}

__global__ void shellsort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper_Results)
{
	// Function body

}

__global__ void shellsort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

__global__ void quicksort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme)
{
	// Function body

}

__global__ void quicksort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper_Results)
{
	// Function body

}

__global__ void quicksort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

