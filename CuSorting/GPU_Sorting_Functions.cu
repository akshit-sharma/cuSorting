#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"
#include "cuSource.h"

__global__ void odd_even_sort_int_xtra(int * d_int, size_t maxLimit, size_t loop)
{
	size_t arrayIndex;
	size_t i, end;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	arrayIndex = arrayIndex * 2;
	
	end = (arrayIndex * loop);

	for (i = loop; i < end; i+=2) {
		arrayIndex = i;
		if (arrayIndex + 1 < maxLimit) {
			if (d_int[arrayIndex] > d_int[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
			}
			__syncthreads();
			arrayIndex += 1;
			if (arrayIndex + 1 < maxLimit) {
				if (d_int[arrayIndex] > d_int[arrayIndex + 1])
				{
					SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
				}
			}
			__syncthreads();
		}
	}
}

__global__ void odd_even_sort_int(int * d_int, size_t maxLimit) {
	size_t arrayIndex;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;
	
	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_int[arrayIndex] > d_int[arrayIndex + 1])
		{
			SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_int[arrayIndex] > d_int[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
			}
		}
		__syncthreads();
	}
}


__global__ void odd_even_sort_llong(long long * d_llong, size_t maxLimit)
{
	size_t arrayIndex;
	int t_llong;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
		{
			SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
			{
				SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
			}
		}
		__syncthreads();
	}
}

__global__ void odd_even_sort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_int(int * d_int, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}


///=============FROM cdpSimpleQuicksort example===============///
__device__ void selection_sort_int(int *data, int left, int right)
{
	for (int i = left; i <= right; ++i)
	{
		int min_val = data[i];
		int min_idx = i;

		// Find the smallest value in the range [left, right].
		for (int j = i + 1; j <= right; ++j)
		{
			int val_j = data[j];

			if (val_j < min_val)
			{
				min_idx = j;
				min_val = val_j;
			}
		}

		// Swap the values.
		if (i != min_idx)
		{
			data[min_idx] = data[i];
			data[i] = min_val;
		}
	}
}

__global__  void quicksort_int(int *data, int left, int right, int depth)
{
/*
	// If we're too deep or there are few elements left, we use an insertion sort...
	if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT)
	{
		selection_sort_int(data, left, right);
		return;
	}

	int *lptr = data + left;
	int *rptr = data + right;
	int  pivot = data[(left + right) / 2];

	// Do the partitioning.
	while (lptr <= rptr)
	{
		// Find the next left- and right-hand values to swap
		int lval = *lptr;
		int rval = *rptr;

		// Move the left pointer as long as the pointed element is smaller than the pivot.
		while (lval < pivot)
		{
			lptr++;
			lval = *lptr;
		}

		// Move the right pointer as long as the pointed element is larger than the pivot.
		while (rval > pivot)
		{
			rptr--;
			rval = *rptr;
		}

		// If the swap points are valid, do the swap!
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	// Now the recursive part
	int nright = rptr - data;
	int nleft = lptr - data;

	// Launch a new block to sort the left part.
	if (left < (rptr - data))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s >>>(data, left, nright, depth + 1);
		hipStreamDestroy(s);
	}

	// Launch a new block to sort the right part.
	if ((lptr - data) < right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s1 >>>(data, nleft, right, depth + 1);
		hipStreamDestroy(s1);
	}
	*/
}

__global__ void quicksort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}
