#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"
#include "cuSource.h"

__global__ void odd_even_sort_int(int * d_int, size_t maxLimit) {
	size_t arrayIndex;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;
	
	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_int[arrayIndex] > d_int[arrayIndex + 1])
		{
			SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_int[arrayIndex] > d_int[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
			}
		}
		__syncthreads();
	}
}


__global__ void odd_even_sort_llong(long long * d_llong, size_t maxLimit)
{
	size_t arrayIndex;
	int t_llong;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
		{
			SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
			{
				SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
			}
		}
		__syncthreads();
	}
}

__global__ void odd_even_sort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_int(int * d_int, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}


///=============FROM cdpSimpleQuicksort example===============///
__device__ void selection_sort(unsigned int *data, int left, int right)
{
	for (int i = left; i <= right; ++i)
	{
		unsigned min_val = data[i];
		int min_idx = i;

		// Find the smallest value in the range [left, right].
		for (int j = i + 1; j <= right; ++j)
		{
			unsigned val_j = data[j];

			if (val_j < min_val)
			{
				min_idx = j;
				min_val = val_j;
			}
		}

		// Swap the values.
		if (i != min_idx)
		{
			data[min_idx] = data[i];
			data[i] = min_val;
		}
	}
}

__global__ void quicksort_int(unsigned int *data, int left, int right, int depth)
{
	// If we're too deep or there are few elements left, we use an insertion sort...
	if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT)
	{
		selection_sort(data, left, right);
		return;
	}

	unsigned int *lptr = data + left;
	unsigned int *rptr = data + right;
	unsigned int  pivot = data[(left + right) / 2];

	// Do the partitioning.
	while (lptr <= rptr)
	{
		// Find the next left- and right-hand values to swap
		unsigned int lval = *lptr;
		unsigned int rval = *rptr;

		// Move the left pointer as long as the pointed element is smaller than the pivot.
		while (lval < pivot)
		{
			lptr++;
			lval = *lptr;
		}

		// Move the right pointer as long as the pointed element is larger than the pivot.
		while (rval > pivot)
		{
			rptr--;
			rval = *rptr;
		}

		// If the swap points are valid, do the swap!
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	// Now the recursive part
	int nright = rptr - data;
	int nleft = lptr - data;

	// Launch a new block to sort the left part.
	if (left < (rptr - data))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s >>>(data, left, nright, depth + 1);
		hipStreamDestroy(s);
	}

	// Launch a new block to sort the right part.
	if ((lptr - data) < right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s1 >>>(data, nleft, right, depth + 1);
		hipStreamDestroy(s1);
	}
}

__global__ void quicksort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}
