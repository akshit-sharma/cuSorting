#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"
#include "cuSource.h"

__global__ void bitonic_sort_int_initMax(int *dev_values)
{
	size_t i;

	i = threadIdx.x + blockDim.x * blockIdx.x;

	dev_values[i] = INT_MAX;
}

__global__ void bitonic_sort_llong_initMax(long long *dev_values)
{
	size_t i;

	i = threadIdx.x + blockDim.x * blockIdx.x;

	dev_values[i] = LONG_MAX;
}

__global__ void bitonic_sort_int(int *dev_values, int j, int k)
{
	unsigned int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i^j;

	if ((ixj) > i) {
		if ((i&k) == 0) {
			if (dev_values[i] > dev_values[ixj]) {
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if ((i&k) != 0) {
			if (dev_values[i] < dev_values[ixj]) {
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}

__global__ void bitonic_sort_llong(long long *dev_values, int j, int k)
{
	unsigned int i, ixj; 
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i^j;

	if ((ixj) > i) {
		if ((i&k) == 0) {
			if (dev_values[i] > dev_values[ixj]) {
				long long temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if ((i&k) != 0) {
			if (dev_values[i] < dev_values[ixj]) {
				long long temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}

__global__ void odd_even_sort_int(int * d_int, size_t maxLimit) {
	size_t arrayIndex;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;
	
	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_int[arrayIndex] > d_int[arrayIndex + 1])
		{
			SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_int[arrayIndex] > d_int[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
			}
		}
		__syncthreads();
	}
}


__global__ void odd_even_sort_llong(long long * d_llong, size_t maxLimit)
{
	size_t arrayIndex;
	int t_llong;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
		{
			SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
			{
				SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
			}
		}
		__syncthreads();
	}
}
__device__ void selection_sort_int(int *data, size_t left, size_t right)
{
	for (int i = left; i <= right; ++i)
	{
		int min_val = data[i];
		size_t min_idx = i;

		// Find the smallest value in the range [left, right].
		for (int j = i + 1; j <= right; ++j)
		{
			int val_j = data[j];

			if (val_j < min_val)
			{
				min_idx = j;
				min_val = val_j;
			}
		}

		// Swap the values.
		if (i != min_idx)
		{
			data[min_idx] = data[i];
			data[i] = min_val;
		}
	}
}

__global__  void quicksort_int(int *data, size_t left, size_t right, size_t depth)
{

	// If we're too deep or there are few elements left, we use an insertion sort...
	if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT)
	{
		selection_sort_int(data, left, right);
		return;
	}

	int *lptr = data + left;
	int *rptr = data + right;
	int  pivot = data[(left + right) / 2];

	// Do the partitioning.
	while (lptr <= rptr)
	{
		// Find the next left- and right-hand values to swap
		int lval = *lptr;
		int rval = *rptr;

		// Move the left pointer as long as the pointed element is smaller than the pivot.
		while (lval < pivot)
		{
			lptr++;
			lval = *lptr;
		}

		// Move the right pointer as long as the pointed element is larger than the pivot.
		while (rval > pivot)
		{
			rptr--;
			rval = *rptr;
		}

		// If the swap points are valid, do the swap!
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	// Now the recursive part
	size_t nright = rptr - data;
	size_t nleft = lptr - data;

	// Launch a new block to sort the left part.
	if (left < (rptr - data))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s >>>(data, left, nright, depth + 1);
		hipStreamDestroy(s);
	}

	// Launch a new block to sort the right part.
	if ((lptr - data) < right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s1 >>>(data, nleft, right, depth + 1);
		hipStreamDestroy(s1);
	}
	
}

__device__ void selection_sort_llong(long long * data, size_t left, size_t right)
{
	for (size_t i = left; i <= right; ++i)
	{
		long long min_val = data[i];
		size_t min_idx = i;

		// Find the smallest value in the range [left, right].
		for (size_t j = i + 1; j <= right; ++j)
		{
			long long val_j = data[j];

			if (val_j < min_val)
			{
				min_idx = j;
				min_val = val_j;
			}
		}

		// Swap the values.
		if (i != min_idx)
		{
			data[min_idx] = data[i];
			data[i] = min_val;
		}
	}
}

__global__ void quicksort_llong(long long * data, size_t left, size_t right, size_t depth)
{

	// If we're too deep or there are few elements left, we use an insertion sort...
	if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT)
	{
		selection_sort_llong(data, left, right);
		return;
	}

	long long *lptr = data + left;
	long long *rptr = data + right;
	long long  pivot = data[(left + right) / 2];

	// Do the partitioning.
	while (lptr <= rptr)
	{
		// Find the next left- and right-hand values to swap
		long long lval = *lptr;
		long long rval = *rptr;

		// Move the left pointer as long as the pointed element is smaller than the pivot.
		while (lval < pivot)
		{
			lptr++;
			lval = *lptr;
		}

		// Move the right pointer as long as the pointed element is larger than the pivot.
		while (rval > pivot)
		{
			rptr--;
			rval = *rptr;
		}

		// If the swap points are valid, do the swap!
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	// Now the recursive part
	size_t nright = rptr - data;
	size_t nleft = lptr - data;

	// Launch a new block to sort the left part.
	if (left < (rptr - data))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		quicksort_llong<<< 1, 1, 0, s >>>(data, left, nright, depth + 1);
		hipStreamDestroy(s);
	}

	// Launch a new block to sort the right part.
	if ((lptr - data) < right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_llong<<< 1, 1, 0, s1 >>>(data, nleft, right, depth + 1);
		hipStreamDestroy(s1);
	}

}
