#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"

__global__ void checkArray(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme, size_t maxLimit)
{
	size_t threadIndex;

	threadIndex = threadIdx.x + blockIdx.x*blockDim.x;

	if (threadIndex + 1 < maxLimit) {
		if (d_PaperIdWrapper_Scheme[threadIndex].paper_id > d_PaperIdWrapper_Scheme[threadIndex+1].paper_id)
		{
			printf("Array not sorted \n %d is before %d \n\n",
				d_PaperIdWrapper_Scheme[threadIndex].paper_id,
				d_PaperIdWrapper_Scheme[threadIndex + 1].paper_id);
		}
	}

}

__global__ void odd_even_sort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper, size_t maxLimit)
{
	size_t threadIndex;
	size_t arrayIndex;
	PaperIdWrapper_Scheme temp;

	threadIndex = threadIdx.x + blockIdx.x*blockDim.x;

	arrayIndex = threadIndex * 2;
	if (arrayIndex+1 < maxLimit) {
		if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
		{
			temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
			temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
			d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
			d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
			d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
			d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
		}
		arrayIndex += 1;
		if (arrayIndex+1 < maxLimit) {
			if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
			{
				temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
				temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
				d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
				d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
				d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
				d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
			}
		}
	}

}

__global__ void odd_even_sort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper)
{
	// Function body

}

__global__ void odd_even_sort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

__global__ void shellsort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme)
{
	// Function body

}

__global__ void shellsort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper_Results)
{
	// Function body

}

__global__ void shellsort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

__global__ void quicksort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme)
{
	// Function body

}

__global__ void quicksort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper_Results)
{
	// Function body

}

__global__ void quicksort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

