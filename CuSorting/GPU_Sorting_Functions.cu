#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"
#include "cuSource.h"

__global__ void odd_even_sort_int(int * d_int, size_t maxLimit) {
	size_t arrayIndex;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;
	
	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_int[arrayIndex] > d_int[arrayIndex + 1])
		{
			SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_int[arrayIndex] > d_int[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
			}
		}
		__syncthreads();
	}
}


__global__ void odd_even_sort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void odd_even_sort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_int(int * d_int, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}


__global__ void quicksort_int(int * d_int, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}
