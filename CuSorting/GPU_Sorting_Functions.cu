#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"

__global__ void checkArray(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme, size_t maxLimit)
{
	size_t threadIndex;

	threadIndex = threadIdx.x + blockIdx.x*blockDim.x;

	if (threadIndex + 1 < maxLimit) {
		if (d_PaperIdWrapper_Scheme[threadIndex].paper_id > d_PaperIdWrapper_Scheme[threadIndex+1].paper_id)
		{
			printf("Array not sorted \n %d is before %d \n\n",
				d_PaperIdWrapper_Scheme[threadIndex].paper_id,
				d_PaperIdWrapper_Scheme[threadIndex + 1].paper_id);
		}
	}

}

__device__ __host__ void printPaperIdCombinationScheme(PaperIdWrapper_Scheme * single_PaperIdWrapper, size_t threadIndex) {
	int paperId;
	SchemeDataStructure * schemeDataStructure;

	paperId = single_PaperIdWrapper->paper_id;
	schemeDataStructure = single_PaperIdWrapper->classPtr;


	printf("paper id %d \n classPtr is %llu threadIdx is %llu \n",
		single_PaperIdWrapper->paper_id,
		single_PaperIdWrapper->classPtr,
		threadIndex
	);

}


__global__ void odd_even_sort_scheme_paperid_swap_edges_only(PaperIdWrapper_Scheme * d_PaperIdWrapper, size_t maxLimit)
{
	size_t threadIndex;
	size_t arrayIndex;
	PaperIdWrapper_Scheme temp;
	size_t internal_max;

	threadIndex = (threadIdx.x + 1) * blockDim.x;

	arrayIndex = threadIndex * 2;

	//printf("kernel called \n");

	if (arrayIndex + 1 < maxLimit) {
		if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
		{
			//			printf("\nbefore even swap from gpu \n");
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex+1], threadIndex);
			temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
			temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
			d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
			d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
			d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
			d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
			//			printf("\nafter even swap from gpu \n");
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
			//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
			{
				//				printf("before odd swap \n");
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
				temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
				temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
				d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
				d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
				d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
				d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
				//				printf("after odd swap \n");
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
				//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
			}
		}
		__syncthreads();
	}

}


__global__ void odd_even_sort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper, size_t maxLimit)
{
	size_t threadIndex;
	size_t arrayIndex;
	PaperIdWrapper_Scheme temp;
	size_t internal_max;

	threadIndex = threadIdx.x + blockIdx.x*blockDim.x;
	
	internal_max = (blockIdx.x + 1) * blockDim.x;

	arrayIndex = threadIndex * 2;

	//printf("kernel called \n");

	if (arrayIndex+1 < maxLimit && arrayIndex+1 < internal_max) {
		if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
		{
//			printf("\nbefore even swap from gpu \n");
//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex+1], threadIndex);
			temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
			temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
			d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
			d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
			d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
			d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
//			printf("\nafter even swap from gpu \n");
//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
//			printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex+1 < maxLimit && arrayIndex + 1 < internal_max) {
			if (d_PaperIdWrapper[arrayIndex].paper_id > d_PaperIdWrapper[arrayIndex + 1].paper_id)
			{
//				printf("before odd swap \n");
//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
				temp.paper_id = d_PaperIdWrapper[arrayIndex].paper_id;
				temp.classPtr = d_PaperIdWrapper[arrayIndex].classPtr;
				d_PaperIdWrapper[arrayIndex].paper_id = d_PaperIdWrapper[arrayIndex + 1].paper_id;
				d_PaperIdWrapper[arrayIndex].classPtr = d_PaperIdWrapper[arrayIndex + 1].classPtr;
				d_PaperIdWrapper[arrayIndex + 1].paper_id = temp.paper_id;
				d_PaperIdWrapper[arrayIndex + 1].classPtr = temp.classPtr;
//				printf("after odd swap \n");
//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex], threadIndex);
//				printPaperIdCombinationScheme(&d_PaperIdWrapper[arrayIndex + 1], threadIndex);
			}
		}
		__syncthreads();
	}

}

__global__ void odd_even_sort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper)
{
	// Function body

}

__global__ void odd_even_sort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

__global__ void shellsort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme)
{
	// Function body

}

__global__ void shellsort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper_Results)
{
	// Function body

}

__global__ void shellsort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

__global__ void quicksort_scheme_paperid(PaperIdWrapper_Scheme * d_PaperIdWrapper_Scheme)
{
	// Function body

}

__global__ void quicksort_results_paperid(PaperIdWrapper_Results * d_PaperIdWrapper_Results)
{
	// Function body

}

__global__ void quicksort_results_rollnumber(RollNumberWrapper_Results * d_results_rollnumberWrapper_Results)
{
	// Function body

}

