#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"
#include "cuSource.h"

__global__ void odd_even_sort_int_xtra(int * d_int, size_t maxLimit, size_t num_arr, size_t arr_size)
{
	size_t arrayIndex;
	size_t arr_end;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	arrayIndex = arrayIndex * 2;

		if (((arrayIndex + 1) < maxLimit) && ((arrayIndex + 1) % arr_size != 0)) {
			if (d_int[arrayIndex] == 0) {
				printf("0 at %llu\n", arrayIndex);
			}
			if (d_int[arrayIndex + 1] == 0) {
				printf("0 at %llu\n", arrayIndex + 1);
			}
			if (d_int[arrayIndex] > d_int[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
			}
			__syncthreads();
			arrayIndex += 1;
			if (((arrayIndex + 1) < maxLimit) && ((arrayIndex + 1) % arr_size != 0)) {
				if (d_int[arrayIndex] == 0) {
					printf("0 at %llu\n", arrayIndex);
				}
				if (d_int[arrayIndex + 1] == 0) {
					printf("0 at %llu\n", arrayIndex + 1);
				}
				if (d_int[arrayIndex] > d_int[arrayIndex + 1])
				{
					SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
				}
			}
			__syncthreads();
		}

}

__global__ void odd_even_sort_int(int * d_int, size_t maxLimit) {
	size_t arrayIndex;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;
	
	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_int[arrayIndex] > d_int[arrayIndex + 1])
		{
			SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_int[arrayIndex] > d_int[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_int);
			}
		}
		__syncthreads();
	}
}


__global__ void odd_even_sort_llong(long long * d_llong, size_t maxLimit)
{
	size_t arrayIndex;
	int t_llong;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
		{
			SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_llong[arrayIndex] > d_llong[arrayIndex + 1])
			{
				SWAP(t_llong, arrayIndex, arrayIndex + 1, d_llong);
			}
		}
		__syncthreads();
	}
}

__global__ void odd_even_sort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_int(int * d_int, size_t maxLimit, size_t num_arr, size_t arr_size, int * d_xtra_int)
{
	size_t arrayIndex;
	int t_int;
	size_t calcIndex;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	calcIndex = ((arrayIndex % num_arr) * arr_size) + (arrayIndex / num_arr);

	if (arrayIndex >= maxLimit)
		return;

	d_xtra_int[calcIndex] = d_int[arrayIndex];

	
	if (d_xtra_int[calcIndex] == 0) {
		printf("Value of %llu is %d is stored as %d at %llu\n", arrayIndex, d_int[arrayIndex], d_xtra_int[calcIndex], calcIndex);
	}
	
	if (d_int[arrayIndex] == 0) {
		printf("Value of %llu is %d is stored as %d at %llu\n", arrayIndex, d_int[arrayIndex], d_xtra_int[calcIndex], calcIndex);
	}

	if (arrayIndex / 2 == 131010) {
		printf("Value of %llu is %d is stored as %d at %llu\n", arrayIndex, d_int[arrayIndex], d_xtra_int[calcIndex], calcIndex);
	}
	if ((calcIndex == 262020) || (calcIndex == 262021)) {
		printf("Value of %llu is %d is stored as %d at %llu\n", arrayIndex, d_int[arrayIndex], d_xtra_int[calcIndex], calcIndex);
	}
	
}

__global__ void shellsort_int_back(int * d_int, size_t maxLimit, size_t num_arr, size_t arr_size, int * d_xtra_int)
{
	size_t arrayIndex;
	int t_int;
	size_t calcIndex;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;

	calcIndex = ((arrayIndex % num_arr) * arr_size) + (arrayIndex / num_arr);

	if (arrayIndex >= maxLimit)
		return;

	d_int[arrayIndex] = d_xtra_int[calcIndex];

//	if (arrayIndex / 2 == 1) {
//		printf("Value of %llu is %d is stored as %d at %llu\n", arrayIndex, d_int[arrayIndex], d_xtra_int[calcIndex], calcIndex);
//	}

}

__global__ void shellsort_llong(long long * d_llong, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}


///=============FROM cdpSimpleQuicksort example===============///
__device__ void selection_sort_int(int *data, size_t left, size_t right)
{
	for (int i = left; i <= right; ++i)
	{
		int min_val = data[i];
		size_t min_idx = i;

		// Find the smallest value in the range [left, right].
		for (int j = i + 1; j <= right; ++j)
		{
			int val_j = data[j];

			if (val_j < min_val)
			{
				min_idx = j;
				min_val = val_j;
			}
		}

		// Swap the values.
		if (i != min_idx)
		{
			data[min_idx] = data[i];
			data[i] = min_val;
		}
	}
}

__global__  void quicksort_int(int *data, size_t left, size_t right, size_t depth)
{

	// If we're too deep or there are few elements left, we use an insertion sort...
	if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT)
	{
		selection_sort_int(data, left, right);
		return;
	}

	int *lptr = data + left;
	int *rptr = data + right;
	int  pivot = data[(left + right) / 2];

	// Do the partitioning.
	while (lptr <= rptr)
	{
		// Find the next left- and right-hand values to swap
		int lval = *lptr;
		int rval = *rptr;

		// Move the left pointer as long as the pointed element is smaller than the pivot.
		while (lval < pivot)
		{
			lptr++;
			lval = *lptr;
		}

		// Move the right pointer as long as the pointed element is larger than the pivot.
		while (rval > pivot)
		{
			rptr--;
			rval = *rptr;
		}

		// If the swap points are valid, do the swap!
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	// Now the recursive part
	size_t nright = rptr - data;
	size_t nleft = lptr - data;

	// Launch a new block to sort the left part.
	if (left < (rptr - data))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s >>>(data, left, nright, depth + 1);
		hipStreamDestroy(s);
	}

	// Launch a new block to sort the right part.
	if ((lptr - data) < right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_int<<< 1, 1, 0, s1 >>>(data, nleft, right, depth + 1);
		hipStreamDestroy(s1);
	}
	
}


///=============modified cdpSimpleQuicksort example===============///
__device__ void selection_sort_llong(long long * data, size_t left, size_t right)
{
	for (size_t i = left; i <= right; ++i)
	{
		long long min_val = data[i];
		size_t min_idx = i;

		// Find the smallest value in the range [left, right].
		for (size_t j = i + 1; j <= right; ++j)
		{
			long long val_j = data[j];

			if (val_j < min_val)
			{
				min_idx = j;
				min_val = val_j;
			}
		}

		// Swap the values.
		if (i != min_idx)
		{
			data[min_idx] = data[i];
			data[i] = min_val;
		}
	}
}

__global__ void quicksort_llong(long long * data, size_t left, size_t right, size_t depth)
{

	// If we're too deep or there are few elements left, we use an insertion sort...
	if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT)
	{
		selection_sort_llong(data, left, right);
		return;
	}

	long long *lptr = data + left;
	long long *rptr = data + right;
	long long  pivot = data[(left + right) / 2];

	// Do the partitioning.
	while (lptr <= rptr)
	{
		// Find the next left- and right-hand values to swap
		long long lval = *lptr;
		long long rval = *rptr;

		// Move the left pointer as long as the pointed element is smaller than the pivot.
		while (lval < pivot)
		{
			lptr++;
			lval = *lptr;
		}

		// Move the right pointer as long as the pointed element is larger than the pivot.
		while (rval > pivot)
		{
			rptr--;
			rval = *rptr;
		}

		// If the swap points are valid, do the swap!
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	// Now the recursive part
	size_t nright = rptr - data;
	size_t nleft = lptr - data;

	// Launch a new block to sort the left part.
	if (left < (rptr - data))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		quicksort_llong<<< 1, 1, 0, s >>>(data, left, nright, depth + 1);
		hipStreamDestroy(s);
	}

	// Launch a new block to sort the right part.
	if ((lptr - data) < right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_llong<<< 1, 1, 0, s1 >>>(data, nleft, right, depth + 1);
		hipStreamDestroy(s1);
	}

}


__global__ void quicksort_string(std::string * d_string, size_t maxLimit)
{
	//Function body

}
