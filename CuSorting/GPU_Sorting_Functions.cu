#include "hip/hip_runtime.h"
#include "GlobalHeader.h"
#include ""
#include "GPU_Sorting_Functions.cuh"
#include "cuRDStruct.h"
#include "cuSDStruct.h"
#include "SchemeDataStructure.h"
#include "cuSource.h"

__global__ void odd_even_sort_scheme_paperid(int * d_paper_id, size_t maxLimit) {
	size_t arrayIndex;
	int t_int;

	arrayIndex = threadIdx.x + blockIdx.x*blockDim.x;
	
	arrayIndex = arrayIndex * 2;

	if (arrayIndex + 1 < maxLimit) {
		if (d_paper_id[arrayIndex] > d_paper_id[arrayIndex + 1])
		{
			SWAP(t_int, arrayIndex, arrayIndex + 1, d_paper_id);
		}
		__syncthreads();
		arrayIndex += 1;
		if (arrayIndex + 1 < maxLimit) {
			if (d_paper_id[arrayIndex] > d_paper_id[arrayIndex + 1])
			{
				SWAP(t_int, arrayIndex, arrayIndex + 1, d_paper_id);
			}
		}
		__syncthreads();
	}
}


__global__ void odd_even_sort_scheme_instiname(std::string * d_institutionNameWrapper_Scheme, size_t maxLimit)
{
	//Function body

}

__global__ void odd_even_sort_scheme_subjname(std::string * d_subjectNameWrapper_Scheme, size_t maxLimit)
{
	//Function body

}

__global__ void odd_even_sort_results_paperid(int * d_int, size_t maxLimit)
{
	//Function body

}

__global__ void odd_even_sort_results_rollnumber(long * d_results_rollnumberWrapper_Results, size_t maxLimit)
{
	//Function body

}

__global__ void odd_even_sort_results_name(std::string * d_nameWrapper_Results, size_t maxLimit)
{
	//Function body

}


__global__ void shellsort_scheme_instiname(std::string * d_institutionNameWrapper_Scheme, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_scheme_subjname(std::string * d_subjectNameWrapper_Scheme, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_scheme_paperid(int * d_PaperId, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_results_paperid(int * d_int, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_results_rollnumber(long * d_results_rollnumberWrapper_Results, size_t maxLimit)
{
	//Function body

}

__global__ void shellsort_results_name(std::string * d_nameWrapper_Results, size_t maxLimit)
{
	//Function body

}


__global__ void quicksort_scheme_instiname(std::string * d_institutionNameWrapper_Scheme, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_scheme_subjname(std::string * d_subjectNameWrapper_Scheme, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_scheme_paperid(int * d_PaperId, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_results_paperid(int * d_int, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_results_rollnumber(long * d_results_rollnumberWrapper_Results, size_t maxLimit)
{
	//Function body

}

__global__ void quicksort_results_name(std::string * d_nameWrapper_Results, size_t maxLimit)
{
	//Function body

}



